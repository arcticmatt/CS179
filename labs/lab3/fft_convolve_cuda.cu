#include "hip/hip_runtime.h"
/* CUDA blur
 * Kevin Yuh, 2014 */

#include <cstdio>
#include <cmath>
#include <climits>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_convolve_cuda.cuh"


/*
Atomic-max function. You may find it useful for normalization.

We haven't really talked about this yet, but __device__ functions not
only are run on the GPU, but are called from within a kernel.

Source:
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}



__global__
void
cudaProdScaleKernel(const hipfftComplex *raw_data, const hipfftComplex *impulse_v,
    hipfftComplex *out_data,
    int padded_length) {


    /* DONE: Implement the point-wise multiplication and scaling for the
    FFT'd input and impulse response.

    Recall that these are complex numbers, so you'll need to use the
    appropriate rule for multiplying them.

    Also remember to scale by the padded length of the signal
    (see the notes for Question 1).

    As in Assignment 1 and Week 1, remember to make your implementation
    resilient to varying numbers of threads.

    */

    // Get current thread's index.
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // For complex number multiplication.
    float a, b, c, d;

    while (thread_index < padded_length) {
        // Point-wise multiplication and scale with real components.
         a = raw_data[thread_index].x;
         b = raw_data[thread_index].y;
         c = impulse_v[thread_index].x;
         d = impulse_v[thread_index].y;
         out_data[thread_index].x = (a * c - b * d) / padded_length;
         out_data[thread_index].y = (a * d + b * c) / padded_length;

        // Update thread_index.
        thread_index += blockDim.x * gridDim.x;
    }
}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* TODO 2: Implement the maximum-finding and subsequent
    normalization (dividing by maximum).

    There are many ways to do this reduction, and some methods
    have much better performance than others.

    For this section: Please explain your approach to the reduction,
    including why you chose the optimizations you did
    (especially as they relate to GPU hardware).

    You'll likely find the above atomicMax function helpful.
    (CUDA's atomicMax function doesn't work for floating-point values.)
    It's based on two principles:
        1) From Week 2, any atomic function can be implemented using
        atomic compare-and-swap.
        2) One can "represent" floating-point values as integers in
        a way that preserves comparison, if the sign of the two
        values is the same. (see http://stackoverflow.com/questions/
        29596797/can-the-return-value-of-float-as-int-be-used-to-
        compare-float-in-cuda)

    */

    // Size determined from third parameter in cudaCallMaximumKernel.
    extern __shared__ float partial_outputs[];

    // Get current thread's index.
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    float thread_max = INT_MIN;
    while (thread_index < padded_length) {
        // Find the maximum MAGNITUDE (take abs value) for this thread.
        thread_max = max(thread_max, fabs(out_data[thread_index].x));

        thread_index += blockDim.x * gridDim.x;
    }

    partial_outputs[threadIdx.x] = thread_max;

    // Make sure all threads in block finish before continuing.
    __syncthreads();

    // Use the first thread in the block to calculate the block's
    // max.
    if (threadIdx.x == 0) {
        float block_max = INT_MIN;

        for (uint thread_idx = 0; thread_idx < blockDim.x; ++thread_idx) {
            block_max = max(block_max, partial_outputs[thread_idx]);
        }

        // Now we take the max with the output.
        atomicMax(max_abs_val, block_max);
    }
}

__global__
void
cudaDivideKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    /* DONE: Implement the division kernel. Divide all
    data by the value pointed to by max_abs_val.

    This kernel should be quite short.
    */

    // Get current thread's index.
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    while (thread_index < padded_length) {
        // Perform the division.
        out_data[thread_index].x /= *max_abs_val;
        out_data[thread_index].y /= *max_abs_val;

        // Update thread_index.
        thread_index += blockDim.x * gridDim.x;
    }
}


void cudaCallProdScaleKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const hipfftComplex *raw_data,
        const hipfftComplex *impulse_v,
        hipfftComplex *out_data,
        const unsigned int padded_length) {

    /* DONE: Call the element-wise product and scaling kernel. */
    cudaProdScaleKernel<<<blocks, threadsPerBlock>>>(raw_data, impulse_v,
            out_data, padded_length);
}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {


    /* DONE: Call the max-finding kernel. */
    cudaMaximumKernel<<<blocks, threadsPerBlock,
        threadsPerBlock * sizeof(float)>>>(out_data, max_abs_val, padded_length);
}


void cudaCallDivideKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {

    /* DONE: Call the division kernel. */
    cudaDivideKernel<<<blocks, threadsPerBlock>>>(out_data, max_abs_val,
            padded_length);
}
