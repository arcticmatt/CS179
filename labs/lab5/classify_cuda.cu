#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "classify_cuda.cuh"

/*
 * Arguments:
 * data: Memory that contains both the review LSA coefficients and the labels.
 *       Format decided by implementation of classify.
 * batch_size: Size of mini-batch, how many elements to process at once
 * step_size: Step size for gradient descent. Tune this as needed. 1.0 is sane
 *            default.
 * weights: Pointer to weights vector of length REVIEW_DIM.
 * errors: Pointer to a single float used to describe the error for the batch.
 *         An output variable for the kernel. The kernel can either write the
 *         value of loss function over the batch or the misclassification rate
 *         in the batch to errors.
 */
__global__
void trainLogRegKernel(
    float *data,
    int batch_size,
    int step_size,
	float *weights,
    float *errors)
{
    // TODO: write me
}

/*
 * All parameters have the same meaning as in docstring for trainLogRegKernel.
 * Notably, cudaClassify returns a float that quantifies the error in the
 * minibatch. This error should go down as more training occurs.
 */
float cudaClassify(
    float *data,
    int batch_size, 
    float step_size,
    float *weights, 
    hipStream_t stream)
{
    int block_size = (batch_size < 1024) ? batch_size : 1024;

    // grid_size = CEIL(batch_size / block_size)
    int grid_size = (batch_size + block_size - 1) / block_size;
    int shmem_bytes = 0;

    float *d_errors;
    hipMalloc(&d_errors, sizeof(float));
    hipMemset(d_errors, 0, sizeof(float));

    trainLogRegKernel<<<grid_size, block_size, shmem_bytes, stream>>>(
        data,
        batch_size,
        step_size,
        weights,
        d_errors);

    float h_errors = -1.0;
    hipMemcpy(&h_errors, d_errors, sizeof(float), hipMemcpyDefault);
    hipFree(d_errors);
    return h_errors;
}
