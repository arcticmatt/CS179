/* CUDA finite difference wave equation solver, written by
 * Jeff Amelang, 2012
 *
 * Modified by Kevin Yuh, 2013-14 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "Cuda1DFDWave_cuda.cuh"


/* TODO: You'll need a kernel here, as well as any helper functions
to call it */